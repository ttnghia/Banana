#include "hip/hip_runtime.h"
//-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+
//-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+
//
//  Copyright (c) 2017 by
//       __      _     _         _____
//    /\ \ \__ _| |__ (_) __ _  /__   \_ __ _   _  ___  _ __   __ _
//   /  \/ / _` | '_ \| |/ _` |   / /\/ '__| | | |/ _ \| '_ \ / _` |
//  / /\  / (_| | | | | | (_| |  / /  | |  | |_| | (_) | | | | (_| |
//  \_\ \/ \__, |_| |_|_|\__,_|  \/   |_|   \__,_|\___/|_| |_|\__, |
//         |___/                                              |___/
//
//  <nghiatruong.vn@gmail.com>
//  All rights reserved.
//
//-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+
//-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "Helpers.cuh"
#include "Common.cuh"
#include "Random.cuh"

//-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+
rtDeclareVariable( float3, shading_normal, attribute shading_normal, );
rtDeclareVariable( float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable( float3, front_hit_point, attribute front_hit_point, );
rtDeclareVariable( float3, texcoord, attribute texcoord, );

rtDeclareVariable(optix::Ray, ray,   rtCurrentRay, );
rtDeclareVariable(PerRayData_Radiance, prd_radiance, rtPayload, );

rtTextureSampler<float4, 2> Kd_map;
rtDeclareVariable( float2, Kd_map_scale, , );

RT_PROGRAM void closest_hit_radiance()
{
    const float3 world_shading_normal   = optix::normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
    const float3 world_geometric_normal = optix::normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
    const float3 ffnormal = optix::faceforward( world_shading_normal, -ray.direction, world_geometric_normal );

    const float z1 = rnd( prd_radiance.seed );
    const float z2 = rnd( prd_radiance.seed );

    float3 w_in;
    optix::cosine_sample_hemisphere( z1, z2, w_in );
    const optix::Onb onb( ffnormal );
    onb.inverse_transform( w_in );
    const float3 fhp = rtTransformPoint( RT_OBJECT_TO_WORLD, front_hit_point );

    prd_radiance.origin = front_hit_point;
    prd_radiance.direction = w_in;

    const float3 Kd = make_float3( tex2D( Kd_map, texcoord.x / Kd_map_scale.x, texcoord.y / Kd_map_scale.y ) );
    prd_radiance.reflectance *= Kd;

}

