#include "hip/hip_runtime.h"
//-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+
//-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+
//
//  Copyright (c) 2017 by
//       __      _     _         _____
//    /\ \ \__ _| |__ (_) __ _  /__   \_ __ _   _  ___  _ __   __ _
//   /  \/ / _` | '_ \| |/ _` |   / /\/ '__| | | |/ _ \| '_ \ / _` |
//  / /\  / (_| | | | | | (_| |  / /  | |  | |_| | (_) | | | | (_| |
//  \_\ \/ \__, |_| |_|_|\__,_|  \/   |_|   \__,_|\___/|_| |_|\__, |
//         |___/                                              |___/
//
//  <nghiatruong.vn@gmail.com>
//  All rights reserved.
//
//-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+
//-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "Common.cuh"
#include "Random.cuh"

//-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(float3, front_hit_point, attribute front_hit_point, );
rtDeclareVariable(float3, back_hit_point, attribute back_hit_point, );

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );

rtDeclareVariable(float,        refraction_index, , );
rtDeclareVariable(float3,       refraction_color, , );
rtDeclareVariable(float3,       reflection_color, , );

// Transmittance at unit distance ( a color with each channel in [0,1] range)
rtDeclareVariable(float3,       unit_transmittance, , );

rtDeclareVariable(PerRayData_Radiance, prd_radiance, rtPayload, );

// -----------------------------------------------------------------------------

static __device__ __inline__ float fresnel( float cos_theta_i, float cos_theta_t, float eta )
{
    const float rs = ( cos_theta_i - cos_theta_t*eta ) /
                     ( cos_theta_i + eta*cos_theta_t );
    const float rp = ( cos_theta_i*eta - cos_theta_t ) /
                     ( cos_theta_i*eta + cos_theta_t );

    return 0.5f * ( rs*rs + rp*rp );
}

static __device__ __inline__ float3 logf( float3 v )
{
    return make_float3( logf(v.x), logf(v.y), logf(v.z) );
}

// -----------------------------------------------------------------------------

RT_PROGRAM void closest_hit_radiance()
{
    const float3 w_out = -ray.direction;
    float3 normal = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
    float cos_theta_i = optix::dot( w_out, normal );

    float eta;
    float3 transmittance = make_float3( 1.0f );
    if( cos_theta_i > 0.0f ) {
        // Ray is entering
        eta = refraction_index;  // Note: does not handle nested dielectrics
    } else {
        // Ray is exiting; apply Beer's Law.
        // This is derived in Shirley's Fundamentals of Graphics book.
        // The "unit_transmittance" is transmittance at unit distance and must
        // be between 0 and 1, so that log(...) is negative.
        transmittance = optix::expf( logf(unit_transmittance) * t_hit );
        eta         = 1.0f / refraction_index;
        cos_theta_i = -cos_theta_i;
        normal      = -normal;
    }

    float3 w_t;
    const bool tir           = !optix::refract( w_t, -w_out, normal, eta );

    const float cos_theta_t  = -optix::dot( normal, w_t );
    const float R            = tir  ?
                               1.0f :
                               fresnel( cos_theta_i, cos_theta_t, eta );

    // Importance sample the Fresnel term
    const float z = rnd( prd_radiance.seed );
    if( z <= R ) {
        // Reflect
        const float3 w_in = optix::reflect( -w_out, normal );
        const float3 fhp = rtTransformPoint(RT_OBJECT_TO_WORLD, front_hit_point);
        prd_radiance.origin = fhp;
        prd_radiance.direction = w_in;
        prd_radiance.reflectance *= reflection_color*transmittance;
    } else {
        // Refract
        const float3 w_in = w_t;
        const float3 bhp = rtTransformPoint(RT_OBJECT_TO_WORLD, back_hit_point);
        prd_radiance.origin = bhp;
        prd_radiance.direction = w_in;
        prd_radiance.reflectance *= refraction_color*transmittance;
    }

    // Note: we do not trace the ray for the next bounce here, we just set it up for
    // the ray-gen program using per-ray data.

}


