#include "hip/hip_runtime.h"
//-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+
//-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+
//
//  Copyright (c) 2017 by
//       __      _     _         _____
//    /\ \ \__ _| |__ (_) __ _  /__   \_ __ _   _  ___  _ __   __ _
//   /  \/ / _` | '_ \| |/ _` |   / /\/ '__| | | |/ _ \| '_ \ / _` |
//  / /\  / (_| | | | | | (_| |  / /  | |  | |_| | (_) | | | | (_| |
//  \_\ \/ \__, |_| |_|_|\__,_|  \/   |_|   \__,_|\___/|_| |_|\__, |
//         |___/                                              |___/
//
//  <nghiatruong.vn@gmail.com>
//  All rights reserved.
//
//-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+
//-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "Helpers.cuh"
#include "Common.cuh"
#include "Random.cuh"

//-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+
rtDeclareVariable(float3,        eye, , );
rtDeclareVariable(float3,        U, , );
rtDeclareVariable(float3,        V, , );
rtDeclareVariable(float3,        W, , );
rtDeclareVariable(float3,        bad_color, , );
rtDeclareVariable(float,         scene_epsilon, , );
rtDeclareVariable(float3,        cutoff_color, , );
rtDeclareVariable(int,           max_depth, , );
rtBuffer<uchar4, 2>              output_buffer;
rtBuffer<float4, 2>              accum_buffer;
rtDeclareVariable(rtObject,      top_object, , );
rtDeclareVariable(unsigned int,  frame, , );
rtDeclareVariable(uint2,         launch_index, rtLaunchIndex, );


RT_PROGRAM void pinhole_camera()
{

  optix::size_t2 screen = output_buffer.size();
  unsigned int seed = tea<16>(screen.x*launch_index.y+launch_index.x, frame);

  // Subpixel jitter: send the ray through a different position inside the pixel each time,
  // to provide antialiasing.
  float2 subpixel_jitter = frame == 0 ? make_float2( 0.0f ) : make_float2(rnd( seed ) - 0.5f, rnd( seed ) - 0.5f);

  float2 d = (make_float2(launch_index) + subpixel_jitter) / make_float2(screen) * 2.f - 1.f;
  float3 ray_origin = eye;
  float3 ray_direction = optix::normalize(d.x*U + d.y*V + W);

  PerRayData_Radiance prd;
  prd.depth = 0;
  prd.seed = seed;
  prd.done = false;

  // These represent the current shading state and will be set by the closest-hit or miss program

  // attenuation (<= 1) from surface interaction.
  prd.reflectance = make_float3( 1.0f );

  // light from a light source or miss program
  prd.radiance = make_float3( 0.0f );

  // next ray to be traced
  prd.origin = make_float3( 0.0f );
  prd.direction = make_float3( 0.0f );

  float3 result = make_float3( 0.0f );

  // Main render loop. This is not recursive, and for high ray depths
  // will generally perform better than tracing radiance rays recursively
  // in closest hit programs.
  for(;;) {
      optix::Ray ray(ray_origin, ray_direction, /*ray type*/ 0, scene_epsilon );
      rtTrace(top_object, ray, prd);

      result += prd.reflectance * prd.radiance;

      if ( prd.done ) {
          break;
      } else if ( prd.depth >= max_depth ) {
        result += prd.reflectance * cutoff_color;
        break;
      }

      prd.depth++;

      // Update ray data for the next path segment
      ray_origin = prd.origin;
      ray_direction = prd.direction;
  }

  float4 acc_val = accum_buffer[launch_index];
  if( frame > 0 ) {
    acc_val = optix::lerp( acc_val, make_float4( result, 0.f ), 1.0f / static_cast<float>( frame+1 ) );
  } else {
    acc_val = make_float4( result, 0.f );
  }
  output_buffer[launch_index] = make_color( make_float3( acc_val ) );
  accum_buffer[launch_index] = acc_val;
}

RT_PROGRAM void exception()
{
  const unsigned int code = rtGetExceptionCode();
  rtPrintf( "Caught exception 0x%X at launch index (%d,%d)\n", code, launch_index.x, launch_index.y );
  output_buffer[launch_index] = make_color( bad_color );
}

