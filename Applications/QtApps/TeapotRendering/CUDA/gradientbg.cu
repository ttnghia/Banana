#include "hip/hip_runtime.h"
//-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+
//-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+
//
//  Copyright (c) 2017 by
//       __      _     _         _____
//    /\ \ \__ _| |__ (_) __ _  /__   \_ __ _   _  ___  _ __   __ _
//   /  \/ / _` | '_ \| |/ _` |   / /\/ '__| | | |/ _ \| '_ \ / _` |
//  / /\  / (_| | | | | | (_| |  / /  | |  | |_| | (_) | | | | (_| |
//  \_\ \/ \__, |_| |_|_|\__,_|  \/   |_|   \__,_|\___/|_| |_|\__, |
//         |___/                                              |___/
//
//  <nghiatruong.vn@gmail.com>
//  All rights reserved.
//
//-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+
//-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "Common.cuh"

//-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+-+
rtDeclareVariable(int, has_envmap, , );
rtDeclareVariable(float3, background_light, , ); // horizon color
rtDeclareVariable(float3, background_dark, , );  // zenith color
rtDeclareVariable(float3, up, , );               // global up vector

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(PerRayData_Radiance, prd_radiance, rtPayload, );

rtTextureSampler<float4, 2> envmap;

// -----------------------------------------------------------------------------
RT_PROGRAM void miss()
{
    if(has_envmap > 0)
    {
          float theta = atan2f( ray.direction.x, ray.direction.z );
          float phi   = M_PIf * 0.5f -  acosf( ray.direction.y );
          float u     = (theta + M_PIf) * (0.5f * M_1_PIf);
          float v     = 0.5f * ( 1.0f + sin(phi) );
          optix::float3 result = optix::make_float3(tex2D(envmap, u, v));

          prd_radiance.radiance = result;
          prd_radiance.done = true;
    }
    else
    {
        const float t = optix::max(optix::dot(ray.direction, up), 0.0f);
        const float3 result = optix::lerp(background_light, background_dark, t);

        prd_radiance.radiance = result;
        prd_radiance.done = true;
    }
}
